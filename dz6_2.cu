
#include "hip/hip_runtime.h"

#include <cmath>
#include <stdio.h>
#define N 10


__global__ void matrixAdd(const int* A, const
    int* B, int* C)
{
    // ���������� ������� �������� ������� �� GPU
    int i = blockIdx.x;
    int j = threadIdx.x;
    
    C[i * N + j] = A[i*N+j] + B[i * N + j];
   // printf("%d,%d,%d=%d+%d\n", i, j, C[i * N + j], A[i * N + j], B[i * N + j]);
}

int main()
{
    int a[N][N];
    int b[N][N];
    int c[N][N];
    int* ca;
    int* cb;
    int* cc;
    for (int i = 0; i < N * N; ++i) {
        *(*a+i) = 1;
    }

    for (int i = 0; i < N * N; ++i) {
        *(*b+i) = 2;
    }

    hipError_t cudaStatus;

    cudaStatus = hipMalloc((void**)&ca, N*N* sizeof(int));

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        return 1;
    }

    cudaStatus = hipMalloc((void**)&cb, N * N * sizeof(int));

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        return 1;
    }


    cudaStatus = hipMalloc((void**)&cc, N * N * sizeof(int));

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        return 1;
    }





    cudaStatus = hipMemcpy(ca, &a, N*N*sizeof(int), hipMemcpyHostToDevice);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpyHostToDevice failed!");
        return 1;
    }

    cudaStatus = hipMemcpy(cb, &b, N*N*sizeof(int), hipMemcpyHostToDevice);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpyHostToDevice failed!");
        return 1;
    }

    // ������ ������� �������
    // ������ ����
    matrixAdd <<<N,N>>> (ca, cb, cc);

    cudaStatus = hipMemcpy(&c, cc, N*N*sizeof(int), hipMemcpyDeviceToHost);


    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpyDeviceToHost failed!");
        return 1;
    }

    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            printf("%d ", c[i][j]);
        }
        printf("\n");
    }

    hipFree(ca);
    hipFree(cb);
    hipFree(cc);
    return 0;
}