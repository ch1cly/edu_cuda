#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "cmath"
#include <stdio.h>

#define N 10


__global__ void calcf(float* c, const float* a, const float* b)
{
    int i = threadIdx.x;
    c[i] = __fmul_rn(b[i], a[i]);
}

__global__ void calcd(double* c, double * a, double * b)
{
    int i = threadIdx.x;
    c[i] = __dmul_rn(b[i], a[i]);
}


__host__ int kernel1() {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float a[] = { 1,1,1,1,1,1,1,1,1,1 };
    float b[] = { 2,2,2,2,2,2,2,2,2,2 };
    float c[N];
    float* ca, * cb, * cc;

    hipError_t cudaStatus;

    cudaStatus = hipMalloc((void**)&ca, N * sizeof(float));

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }
    cudaStatus = hipMalloc((void**)&cb, N * sizeof(float));

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }
    cudaStatus = hipMalloc((void**)&cc, N * sizeof(float));

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }


    cudaStatus = hipMemcpy(ca, a, N * sizeof(float), hipMemcpyHostToDevice);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }

    cudaStatus = hipMemcpy(cb, b, N * sizeof(float), hipMemcpyHostToDevice);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }

    hipEventRecord(start, 0);

    calcf << <1, N >> > (cc, ca, cb);
    // Copy input vectors from host memory to GPU buffers.


    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float KernelTime;
    hipEventElapsedTime(&KernelTime, start, stop);
    printf("KernelTime float: %.2f milliseconds\n",
        KernelTime);
    cudaStatus = hipMemcpy(c, cc, N * sizeof(float), hipMemcpyDeviceToHost);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    float ansv = 0;
    for (int i = 0; i < N; ++i) {
        ansv += c[i];
    }
    printf("ansv if %f\n", ansv);
    hipFree(ca);
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    return 0;
}


__host__ int kernal2() {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    double a[] = { 1,1,1,1,1,1,1,1,1,1 };
    double b[] = { 2,2,2,2,2,2,2,2,2,2 };
    double c[N];
    double* ca, * cb, * cc;

    hipError_t cudaStatus;

    cudaStatus = hipMalloc((void**)&ca, N * sizeof(double));

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }
    cudaStatus = hipMalloc((void**)&cb, N * sizeof(double));

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }
    cudaStatus = hipMalloc((void**)&cc, N * sizeof(double));

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }


    cudaStatus = hipMemcpy(ca, a, N * sizeof(double), hipMemcpyHostToDevice);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }

    cudaStatus = hipMemcpy(cb, b, N * sizeof(double), hipMemcpyHostToDevice);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }

    hipEventRecord(start, 0);

    calcd << <1, N >> > (cc, ca, cb);
    // Copy input vectors from host memory to GPU buffers.


    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float KernelTime;
    hipEventElapsedTime(&KernelTime, start, stop);
    printf("KernelTime double: %.2f milliseconds\n",
        KernelTime);
    cudaStatus = hipMemcpy(c, cc, N * sizeof(double), hipMemcpyDeviceToHost);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    double ansv = 0;
    for (int i = 0; i < N; ++i) {
        ansv += c[i];
    }
    printf("ansv if %f\n", ansv);
    hipFree(ca);
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

}

int main() {

    kernel1();
    kernal2();
    
    return 0;
}