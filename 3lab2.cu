﻿
#include "hip/hip_runtime.h"

#include <cmath>
#include <stdio.h>
#define N 100


__global__ void integral(double *a)
{
    int i = threadIdx.x;
    a[i] = std::sqrtf(1.0 - double(i) * double(i) / double(N) / double(N));
}

int main()
{
    double a[N] = { 0 };
    double* p_a;

    hipError_t cudaStatus;

    cudaStatus = hipMalloc((void**)&p_a, N * sizeof(double));

   
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        return 1;
    }

    integral <<<1, N >>> (p_a);

    cudaStatus = hipMemcpy(a, p_a, N * sizeof(double), hipMemcpyDeviceToHost);


    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpyDeviceToHost failed!");
        return 1;
    }
    
    double q = 0;
    for (int i = 0; i < N; ++i) {
        q += a[i];
    }
    printf("Pi is %f\n", q*4/N);

    hipFree(p_a);
    return 0;
}
