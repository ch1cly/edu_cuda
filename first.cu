#include "hip/hip_runtime.h"
%%cu
#include <stdio.h>
#include <stdlib.h>
__global__ void add(int *a, int *b, int *c) {
*c = *a + *b;
}
int main() {
int a, b, c;
// host copies of variables a, b & c
int *d_a, *d_b, *d_c;
// device copies of variables a, b & c
int size = sizeof(int);
// Allocate space for device copies of a, b, c
hipMalloc((void **)&d_a, size);
hipMalloc((void **)&d_b, size);
hipMalloc((void **)&d_c, size);
// Setup input values  
c = 0;
a = 3;
b = 5;
// Copy inputs to device
hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
// Launch add() kernel on GPU
add<<<1,1>>>(d_a, d_b, d_c);
// Copy result back to host
hipError_t err = hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
  if(err!=hipSuccess) {
      printf("CUDA error copying to Host: %s\n", hipGetErrorString(err));
  }
printf("result is %d\n",c);
// Cleanup
hipFree(d_a);
hipFree(d_b);
hipFree(d_c);
return 0;
}




%%cu
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include ""

int main() {
hipDeviceProp_t deviceProp;
hipGetDeviceProperties(&deviceProp, 0);
printf("Multiprocessor count: %d\n", deviceProp.multiProcessorCount);
printf("Total global memory : %d MB\n", deviceProp.totalGlobalMem / 1024 / 1024); 
printf("Clock rate: %d\n", deviceProp.clockRate); 
printf("Peak memory clock frequency in kilohertz: %d\n", deviceProp.memoryClockRate); 


printf("Global memory bus width in bits: %d\n", deviceProp.memoryBusWidth);

printf("Conclusion: %s is pretty good for me obv.\n", deviceProp.name); 
return 0;
}
