#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <cmath>
#include <stdio.h>


#define N 100


__global__ void dzeta(float *s,float*c)
{

    c[threadIdx.x] = 1.f / powf(float(threadIdx.x + 1), *s);


}


int main()
{

    float x = 2;
    float c[N] = { 0 };

    
    float* dev_x = 0;
    float* dev_c = 0;

    hipError_t cudaStatus;

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, N * sizeof(float));



    cudaStatus = hipMalloc((void**)&dev_x, sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        return 3;
    }

    // Copy input vectors from host memory to GPU buffers.
  
    cudaStatus = hipMemcpy(dev_c, c, N * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        return 5;
    }

    cudaStatus = hipMemcpy(dev_x, &x,  sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        return 6;
    }

  
    dzeta << <1, N >> > (dev_x, dev_c);

   
    
    cudaStatus = hipMemcpy(c, dev_c, N * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        return 7;
    }


    double sum = 0;
    for (int i = 0; i < N; ++i) {
       // printf("a = %f\n", a[i]);
        sum += c[i];
    }
    printf("dzeta = %f\n", sum);


    hipFree(dev_c);
   // hipFree(dev_a);
    hipFree(dev_x);



    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 10;
    }



    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 11;
    }

    return 0;
}