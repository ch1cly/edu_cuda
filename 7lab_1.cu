#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>


#define N 100



__global__ void skalarProduct(double*c, const double*a, const double*b)
{
    __shared__ double ash[32];
    __shared__ double bsh[32];
    __shared__ double csh[32];
    // Копирование из глобальной памяти
    if (blockIdx.x * 32 + threadIdx.x < N) {
        ash[threadIdx.x] = a[blockIdx.x * 32 + threadIdx.x];
        bsh[threadIdx.x] = b[blockIdx.x * 32 + threadIdx.x];
    }
    // Синхронизация нитей
    
    csh[threadIdx.x] = ash[threadIdx.x] * bsh[threadIdx.x];

    __syncthreads();

    c[blockIdx.x * 32 + threadIdx.x] = csh[threadIdx.x];


    __syncthreads();

    // Вычисление скалярного произведения
    
}

int main()
{
    
    double a[N];
    double b[N];
    double c[N] = {0};

    for (int i = 0; i < N; ++i) {
        a[i] = 1;
        b[i] = 2;
    }
    b[N - 1] = 3;

    double* dev_a = 0;
    double* dev_b = 0;
    double* dev_c = 0;
    hipError_t cudaStatus;

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, N * sizeof(double));

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        return 1;
    }

    cudaStatus = hipMalloc((void**)&dev_a, N * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        return 2;
    }

    cudaStatus = hipMalloc((void**)&dev_b, N * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        return 3;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, N * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        return 4;
    }

    cudaStatus = hipMemcpy(dev_b, b, N * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        return 5;
    }


    int blockSize, threadNum;

    blockSize = N / 32 + 1;
    // Launch a kernel on the GPU with one thread for each element.


hipEvent_t start, stop;
    float elapsedTime;
    // создаем события
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // запись события
    hipEventRecord(start, 0);

    skalarProduct << <blockSize, 32 >> > (dev_c, dev_a, dev_b);

 hipEventRecord(stop, 0);
    // ожидание завершения работы ядра
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    // вывод информации
    printf("Time spent executing by the GPU: %.2f millseconds\n", elapsedTime);
    // уничтожение события
    hipEventDestroy(start);
    hipEventDestroy(stop);

    cudaStatus = hipMemcpy(c, dev_c, N * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        return 6;
    }

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        return 7;
    }

    double sum = 0;
    for (int i = 0; i < N; ++i) {
        printf("a = %f\n", a[i]);
        sum += c[i];
    }

    printf("skal product = %f\n",sum);


    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        return 8;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, N * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        return 9;
    }

    

    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 10;
    }

   

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 11;
    }

    return 0;
}

