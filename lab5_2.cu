#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "cmath"
#include <stdio.h>

#define N 10


__global__ void calc(float *c,const float* a,const float* b)
{
    int i = threadIdx.x;
    c[i] = a[i]*b[i];
}

int main(){

    float a[] = {1,1,1,1,1,1,1,1,1,1};
    float b[] = {2,2,2,2,2,2,2,2,2,2};
    float c [N];
    float* ca,*cb,*cc;

    hipError_t cudaStatus;

    cudaStatus = hipMalloc((void**)&ca, N * sizeof(float));
    
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }
    cudaStatus = hipMalloc((void**)&cb, N * sizeof(float));

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }
    cudaStatus = hipMalloc((void**)&cc, N * sizeof(float));

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }


    cudaStatus = hipMemcpy(ca, a, N * sizeof(float), hipMemcpyHostToDevice);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }

    cudaStatus = hipMemcpy(cb, b, N * sizeof(float), hipMemcpyHostToDevice);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }

 

    calc <<<1, N >> > (cc,ca,cb);
    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(c, cc, N * sizeof(float), hipMemcpyDeviceToHost);
    
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }
    
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    float ansv=0;
    for (int i = 0; i < N; ++i) {
        ansv += c[i];
    }

    printf("ansv if %f\n",ansv);
    hipFree(ca);

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}
