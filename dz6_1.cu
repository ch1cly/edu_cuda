#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <cmath>

#define BLOCK_SIZE 16
// ���, ������� ����� ����� �������� ������
#define BASE_TYPE double
// ������� ������������ ������
__global__ void matrixMult(const BASE_TYPE* A, const
	BASE_TYPE* B, BASE_TYPE* C, int Acols, int Bcols)
{
	int i0 = Acols * (blockDim.y * blockIdx.y +
		threadIdx.y);
	int j0 = blockDim.x * blockIdx.x + threadIdx.x;
	BASE_TYPE sum = 0;

		for (int k = 0; k < Acols; k++)
			sum += A[i0 + k] * B[k * Bcols + j0];

	int ind = Bcols * (blockDim.y * blockIdx.y +
		threadIdx.y) + blockDim.x * blockIdx.x + threadIdx.x;
	C[ind] = sum;
}
int toMultiple(int a, int b) {
	int mod = a % b;
	if (mod != 0) {
		mod = b - mod;
		return a + mod;
	}
	return a;
}
int main()
{
	//start, stop - for Kernel time
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// ���������� ����� � �������� �������
	int Arows = 100;
	int Acols = 200;
	int Brows = Acols;
	int Bcols = 150;
	Arows = toMultiple(Arows, BLOCK_SIZE);
	printf("Arows = %d\n", Arows);
	Acols = toMultiple(Acols, BLOCK_SIZE);
	printf("Acols = %d\n", Acols);
	Brows = toMultiple(Brows, BLOCK_SIZE);
	printf("Brows = %d\n", Brows);
	Bcols = toMultiple(Bcols, BLOCK_SIZE);
	printf("Bcols = %d\n", Bcols);
	size_t Asize = Arows * Acols * sizeof(BASE_TYPE);
	size_t Bsize = Brows * Bcols * sizeof(BASE_TYPE);
	size_t Csize = Arows * Bcols * sizeof(BASE_TYPE);
	BASE_TYPE* h_A = (BASE_TYPE*)malloc(Asize);
	BASE_TYPE* h_B = (BASE_TYPE*)malloc(Bsize);
	BASE_TYPE* h_AB = (BASE_TYPE*)malloc(Csize);
	BASE_TYPE* h_BA = (BASE_TYPE*)malloc(Csize);

	for (int i = 0; i < Arows * Acols; ++i) {
		h_A[i] = rand() / (BASE_TYPE)RAND_MAX;
	}
	for (int i = 0; i < Brows * Bcols; ++i) {
		h_B[i] = rand() / (BASE_TYPE)RAND_MAX;
	}
	BASE_TYPE* d_A = NULL;
	hipMalloc((void**)&d_A, Asize);
	BASE_TYPE* d_B = NULL;
	hipMalloc((void**)&d_B, Bsize);
	BASE_TYPE* d_AB = NULL;
	hipMalloc((void**)&d_AB, Csize);

	BASE_TYPE* d_BA = NULL;
	hipMalloc((void**)&d_BA, Csize);

	hipMemcpy(d_A, h_A, Asize, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, Bsize, hipMemcpyHostToDevice);
	dim3 threadsPerBlock = dim3(BLOCK_SIZE, BLOCK_SIZE);
	dim3 blocksPerGrid = dim3(Bcols / BLOCK_SIZE, Arows /
		BLOCK_SIZE);


	hipEventRecord(start, 0);
	matrixMult << <blocksPerGrid, threadsPerBlock >> > (d_A, d_B, d_AB, Acols, Bcols);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float KernelTime;
	hipEventElapsedTime(&KernelTime, start, stop);
	printf("KernelTime: %.2f milliseconds\n",
		KernelTime);
	hipMemcpy(h_AB, d_AB, Csize, hipMemcpyDeviceToHost);


	hipEventRecord(start, 0);
	matrixMult << <blocksPerGrid, threadsPerBlock >> > (d_B, d_A, d_BA, Acols, Bcols);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float KernelTime1;
	hipEventElapsedTime(&KernelTime1, start, stop);
	printf("KernelTime: %.2f milliseconds\n",
		KernelTime1);
	hipMemcpy(h_BA, d_BA, Csize, hipMemcpyDeviceToHost);

	bool b = true;
	for (int i = 0; i < Csize; ++i) {
		if (abs(h_AB[i] - h_BA[i]) > 1e-9) {
			b = false;
			break;
		}
	}
	if (!b) {
		printf("not comm\n");
	}
	else {
		printf("comm\n");
	}
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_AB);
	hipFree(d_BA);
	free(h_A);
	free(h_B);
	free(h_AB);
	free(h_BA);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	return 0;
}