#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <iostream>

__global__ void add(int* a, int* b, int* c) {
    *c = *a + *b;
}

int main() {
    int a, b, c;
    
    // host copies of variables a, b & c
    int* d_a, * d_b, * d_c;
    // device copies of variables a, b & c
    int size = sizeof(int);
    
    // Allocate space for device copies of a, b, c
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);
    
    // Setup input values  
    c = 0;
    //a = 3;
    //b = 5;
    std::cout << "Input first figure" << std::endl;
    std::cin >> a;
    std::cout << "Input second figure" << std::endl;
    std::cin >> b;
    
    // Copy inputs to device
    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
    
    // Launch add() kernel on GPU
    add<<<1,1>>> (d_a, d_b, d_c);
    
    // Copy result back to host
    hipError_t err = hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("CUDA error copying to Host: %s\n", hipGetErrorString(err));
    }
    printf("\nresult is %d\n", c);
    
    // Cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}
