#include "hip/hip_runtime.h"

%%cu
#include "hip/hip_runtime.h"
#include ""
#include <cmath>
#include <stdio.h>
#define N 1000


__global__ void integral(double **a)
{
    int x = threadIdx.x;
    int y = blockIdx.x;
    if (x * x + y * y > 1){
        a[x][y] = 0;
    } 
    else{
        a[x][y] = 1;
    }
}

int main()
{
    double a[N][N] = { 0 };
    double** p_a;

    hipError_t cudaStatus;

    cudaStatus = hipMalloc((void**)&p_a, N * sizeof(*double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc p_a failed!");
        return 1;
    }
    for (int i = 0; i < N; ++i) {
        cudaStatus = hipMalloc((void**)&(*p_a+i), N * sizeof(double));
        if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc inside p_a failed!");
        return 1;
      }
    }
   
    

    integral <<<N, N >>> (p_a);

    cudaStatus = hipMemcpy(a, p_a, N * sizeof(*double), hipMemcpyDeviceToHost);


    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpyDeviceToHost failed!");
        return 1;
    }
 
  for (int i = 0; i < N; ++i) {
      cudaStatus = hipMemcpy((*a+i, *p_a+i, N * sizeof(double), hipMemcpyDeviceToHost);


    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpyDeviceToHost inside failed!");
        return 1;
    }
  }
    
    double q = 0;
    for(int j = 0; j < N; ++j){
    for (int i = 0; i < N; ++i) {
        q += a[j][i];
    }
    }
    printf("Pi is %f\n", q*4);

     for (int i = 0; i < N; ++i) {
       hipFree(*p_a+i);
     }

    hipFree(p_a);
    return 0;
}