﻿
#include "hip/hip_runtime.h"

#include <cmath>
#include <stdio.h>
#define N 1000


__device__ bool isInCircle(double *x, double *y)
{
    return (*x) * (*x) + (*y) * (*y) <= 1;
}

__global__ void piCalc(double* p_a) {

    double x = double(blockIdx.x) / N;
    double y = double(threadIdx.x) / N;
    isInCircle(&x, &y) ? p_a[threadIdx.x * N + blockIdx.x] = 1 : p_a[threadIdx.x * N + blockIdx.x] = 0;
}

int main()
{
    double *a = new double[N*N];
    double* p_a;


    hipError_t cudaStatus;

    cudaStatus = hipMalloc((void**)&p_a, N * N * sizeof(double));
    
   
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        return 1;
    }

    piCalc <<< N, N >>> (p_a);

    cudaStatus = hipMemcpy(a, p_a, N * N * sizeof(double), hipMemcpyDeviceToHost);


    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpyDeviceToHost failed!");
        return 1;
    }
    
    double q = 0;
    for (int i = 0; i < N*N; ++i) {
        q += a[i];
    }
    printf("Pi is %f\n", q*4/N/N);
    
    delete a;
    hipFree(p_a);
    return 0;
}
