#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "cmath"
#include <stdio.h>

#define N 100


__global__ void calc(float *c)
{
    int i = threadIdx.x;
    c[i] = __expf(float(i)/float(100));
}

int main(){

    float a[N];
    float* ca;

    hipError_t cudaStatus;

    cudaStatus = hipMalloc((void**)&ca, N * sizeof(float));
    
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }

    calc <<<1, N >> > (ca);
    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(a, ca, N * sizeof(float), hipMemcpyDeviceToHost);
    
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }
    
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    for (int i = 0; i < N; ++i) {
        printf("err is %f\n", abs(exp(float(i)/float(100)) - a[i]));
    }
    hipFree(ca);

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}
